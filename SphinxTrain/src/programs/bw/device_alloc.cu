#include "hip/hip_runtime.h"

#include "device_alloc.h"
#include <sphinxbase/ckd_alloc.h>
#include <cutil.h>

void gauden_dev_free(gauden_dev_t *g) {

    hipFree((void *)g->d_veclen);
    hipFree((void *)g->d_norm);
    
    hipFree((void *)g->d_cb);
    hipFree((void *)g->d_l_cb);
    hipFree((void *)g->d_mixw);
    
    hipFree((void *)g->d_mean_idx);
    hipFree((void *)g->d_mean_buf);

    hipFree((void *)g->d_var_idx);
    hipFree((void *)g->d_var_buf);

    ckd_free((void *)g);
}

gauden_dev_t *gauden_dev_copy(model_inventory_t *inv, state_t *state_seq, uint32 n_state) {

    gauden_dev_t *g;
    uint32 *buf;
    uint32 s;
    
    g = (gauden_dev_t *)ckd_calloc(1, sizeof(gauden_dev_t));
    
    g->n_feat = inv->gauden->n_feat;
    g->n_mgau = inv->gauden->n_mgau;
    g->n_density = inv->gauden->n_density;
    g->n_top = inv->gauden->n_top;
    g->n_cb_inverse = inv->n_cb_inverse;
    g->n_state = n_state;
    
    E_INFO("MICHAL: %u %u %u %u %u %u\n", g->n_feat, g->n_mgau, g->n_density, g->n_top, g->n_cb_inverse, g->n_state);
    
    hipMalloc(&g->d_veclen, g->n_feat * sizeof(uint32));
    hipMalloc(&g->d_norm, g->n_mgau * g->n_feat * g->n_density * sizeof(float32));
    
    hipMalloc(&g->d_cb, g->n_state * sizeof(uint32));
    hipMalloc(&g->d_l_cb, g->n_state * sizeof(uint32));
    hipMalloc(&g->d_mixw, g->n_state * sizeof(uint32));
    
    g->d_mean_buflen = inv->gauden->mean[0][0][g->n_mgau * g->n_feat * g->n_density - 1] - inv->gauden->mean[0][0][0] + inv->gauden->veclen[g->n_feat - 1];
    hipMalloc(&g->d_mean_idx, g->n_mgau * g->n_feat * g->n_density * sizeof(float *));
    hipMalloc(&g->d_mean_buf, g->d_mean_buflen * sizeof(float));

    g->d_var_buflen = inv->gauden->var[0][0][g->n_mgau * g->n_feat * g->n_density - 1] - inv->gauden->var[0][0][0] + inv->gauden->veclen[g->n_feat - 1];
    hipMalloc(&g->d_var_idx, g->n_mgau * g->n_feat * g->n_density * sizeof(float *));
    hipMalloc(&g->d_var_buf, g->d_var_buflen * sizeof(float));
    
    /* veclen, norm, den, den_idx */
    hipMemcpy(g->d_veclen, inv->gauden->veclen, g->n_feat * sizeof(uint32), hipMemcpyHostToDevice);
    hipMemcpy(g->d_norm, inv->gauden->norm[0][0], g->n_mgau * g->n_feat * g->n_density * sizeof(float32), hipMemcpyHostToDevice);
    
    /* state_seq -> d_cb, d_l_cb, d_mixw */
    buf = (uint32 *)ckd_calloc(g->n_state, sizeof(uint32));
    for (s = 0; s < g->n_state; s++) buf[s] = state_seq[s].cb;
    hipMemcpy(g->d_cb, buf, g->n_state * sizeof(uint32), hipMemcpyHostToDevice);
    for (s = 0; s < g->n_state; s++) buf[s] = state_seq[s].l_cb;
    hipMemcpy(g->d_l_cb, buf, g->n_state * sizeof(uint32), hipMemcpyHostToDevice);
    for (s = 0; s < g->n_state; s++) buf[s] = state_seq[s].mixw;
    hipMemcpy(g->d_mixw, buf, g->n_state * sizeof(uint32), hipMemcpyHostToDevice);
    ckd_free((void *)buf);
    
    /* mean, var, feature */
    hipMemcpy(g->d_mean_idx, inv->gauden->mean[0][0], g->n_mgau * g->n_feat * g->n_density * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(g->d_mean_buf, inv->gauden->mean[0][0][0], g->d_mean_buflen * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g->d_var_idx, inv->gauden->var[0][0], g->n_mgau * g->n_feat * g->n_density * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(g->d_var_buf, inv->gauden->var[0][0][0], g->d_var_buflen * sizeof(float), hipMemcpyHostToDevice);

    return g;
}


__global__ void device_init_3d_kernel(char *mem, char ***ref1, char **ref2, size_t elemsize, size_t d1, size_t d2, size_t d3);
__global__ void device_init_4d_kernel(char *mem, char ****ref1, char ***ref2, char **ref3, size_t elemsize, size_t d1, size_t d2, size_t d3, size_t d4);


void *
device_alloc_3d(size_t d1, size_t d2, size_t d3, size_t elemsize)
{
    dim3 bdim(16, 1, 1);
    dim3 gdim(ceil(d1 / (float)bdim.x), 1, 1);

    char ***ref1, **ref2, *mem;

    hipMalloc(&mem, d1 * d2 * d3 * elemsize);
    hipMalloc(&ref2, d1 * d2 * sizeof(void *));
    hipMalloc(&ref1, d1 * sizeof(void **));

    device_init_3d_kernel<<<gdim, bdim>>>(mem, ref1, ref2, elemsize, d1, d2, d3);

    return ref1;
}


void *
device_alloc_4d(size_t d1,
		  size_t d2,
		  size_t d3,
		  size_t d4,
		  size_t elemsize)
{
    dim3 bdim(16, 1, 1);
    dim3 gdim(ceil(d1 / (float)bdim.x), 1, 1);

    char *mem;
    char ****ref1;
    char ***ref2;
    char **ref3;

    hipMalloc(&mem, d1 * d2 * d3 * d4 * elemsize);
    hipMalloc(&ref3, d1 * d2 * d3 * sizeof(void *));
    hipMalloc(&ref2, d1 * d2 * sizeof(void **));
    hipMalloc(&ref1, d1 * sizeof(void ***));
    
    device_init_4d_kernel<<<gdim, bdim>>>(mem, ref1, ref2, ref3, elemsize, d1, d2, d3, d4);
    
    return ref1;
}


void
device_free_3d(void *inptr) {
    void ***ptr = (void ***)inptr;
    if (ptr == NULL)
        return;
    hipFree(ptr[0][0]);
    hipFree(ptr[0]);
    hipFree(ptr);
}


void
device_free_4d(void *inptr) {
    void ****ptr = (void ****)inptr;
    if (ptr == NULL)
        return;
    hipFree(ptr[0][0][0]);
    hipFree(ptr[0][0]);
    hipFree(ptr[0]);
    hipFree(ptr);
}


/* kernels */

__global__ void device_init_3d_kernel(char *mem, char ***ref1, char **ref2, size_t elemsize, size_t d1, size_t d2, size_t d3) {
    int i, j;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d1) return;
    
    ref1[i] = ref2 + i * d2;
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        ref1[i][j] = mem + (i * d2 + j) * d3 * elemsize;
    }
}


__global__ void device_init_4d_kernel(char *mem, char ****ref1, char ***ref2, char **ref3, size_t elemsize, size_t d1, size_t d2, size_t d3, size_t d4) {
    int i, j, k;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d1) return;
    
    ref1[i] = ref2 + i * d2;
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        ref1[i][j] = ref3 + (i * d2 + j) * d3;
    }
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        for (k = 0; k < d3; k++) {
            ref1[i][j][k] = mem + ((i * d2 + j) * d3 + k) * d4 * elemsize;
        }
    }
}


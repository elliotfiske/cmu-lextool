#include "hip/hip_runtime.h"

#include "device_alloc.h"
#include <cutil.h>


__global__ void device_init_3d_kernel(char *mem, char ***ref1, char **ref2, size_t elemsize, size_t d1, size_t d2, size_t d3);
__global__ void device_init_4d_kernel(char *mem, char ****ref1, char ***ref2, char **ref3, size_t elemsize, size_t d1, size_t d2, size_t d3, size_t d4);


void *
device_alloc_3d(size_t d1, size_t d2, size_t d3, size_t elemsize)
{
    dim3 bdim(16, 1, 1);
    dim3 gdim(ceil(d1 / (float)bdim.x), 1, 1);

    char ***ref1, **ref2, *mem;

    hipMalloc(&mem, d1 * d2 * d3 * elemsize);
    hipMalloc(&ref2, d1 * d2 * sizeof(void *));
    hipMalloc(&ref1, d1 * sizeof(void **));

    device_init_3d_kernel<<<gdim, bdim>>>(mem, ref1, ref2, elemsize, d1, d2, d3);

    return ref1;
}


void *
device_alloc_4d(size_t d1,
		  size_t d2,
		  size_t d3,
		  size_t d4,
		  size_t elemsize)
{
    dim3 bdim(16, 1, 1);
    dim3 gdim(ceil(d1 / (float)bdim.x), 1, 1);

    char *mem;
    char ****ref1;
    char ***ref2;
    char **ref3;

    hipMalloc(&mem, d1 * d2 * d3 * d4 * elemsize);
    hipMalloc(&ref3, d1 * d2 * d3 * sizeof(void *));
    hipMalloc(&ref2, d1 * d2 * sizeof(void **));
    hipMalloc(&ref1, d1 * sizeof(void ***));
    
    device_init_4d_kernel<<<gdim, bdim>>>(mem, ref1, ref2, ref3, elemsize, d1, d2, d3, d4);
    
    return ref1;
}


void
device_free_3d(void *inptr) {
    void ***ptr = (void ***)inptr;
    if (ptr == NULL)
        return;
    hipFree(ptr[0][0]);
    hipFree(ptr[0]);
    hipFree(ptr);
}


void
device_free_4d(void *inptr) {
    void ****ptr = (void ****)inptr;
    if (ptr == NULL)
        return;
    hipFree(ptr[0][0][0]);
    hipFree(ptr[0][0]);
    hipFree(ptr[0]);
    hipFree(ptr);
}


/* kernels */

__global__ void device_init_3d_kernel(char *mem, char ***ref1, char **ref2, size_t elemsize, size_t d1, size_t d2, size_t d3) {
    int i, j;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d1) return;
    
    ref1[i] = ref2 + i * d2;
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        ref1[i][j] = mem + (i * d2 + j) * d3 * elemsize;
    }
}


__global__ void device_init_4d_kernel(char *mem, char ****ref1, char ***ref2, char **ref3, size_t elemsize, size_t d1, size_t d2, size_t d3, size_t d4) {
    int i, j, k;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d1) return;
    
    ref1[i] = ref2 + i * d2;
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        ref1[i][j] = ref3 + (i * d2 + j) * d3;
    }
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        for (k = 0; k < d3; k++) {
            ref1[i][j][k] = mem + ((i * d2 + j) * d3 + k) * d4 * elemsize;
        }
    }
}


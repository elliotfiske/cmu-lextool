#include "hip/hip_runtime.h"

#include "device_alloc.h"
#include <sphinxbase/ckd_alloc.h>
#include <cutil.h>


#ifdef STOPWATCH

#include <sys/time.h>

void startTimer(struct timeval *timer){
        gettimeofday(timer, NULL);
}

uint32 stopTimer(struct timeval *timer){
        struct timeval tmp;
        gettimeofday(&tmp, NULL);
        tmp.tv_sec -= timer->tv_sec;
        tmp.tv_usec -= timer->tv_usec;
        if (tmp.tv_usec < 0){
                tmp.tv_usec+=1000000;
                tmp.tv_sec--;
        }
        return (uint32)(tmp.tv_usec + tmp.tv_sec*1000000);
}

#endif


void gauden_dev_free(gauden_dev_t *g) {
#ifdef GAUDEN_DEV
    if (g) {
        hipFree((void *)g->d_veclen);
        hipFree((void *)g->d_norm);
        
        hipFree((void *)g->d_cb);
        hipFree((void *)g->d_l_cb);
        hipFree((void *)g->d_active_states);
        
        hipFree((void *)g->d_mean_idx);
        hipFree((void *)g->d_mean_buf);

        hipFree((void *)g->d_var_idx);
        hipFree((void *)g->d_var_buf);
        
        hipFree((void *)g->d_feature_idx);
        hipFree((void *)g->d_feature_buf);
        
        hipFree((void *)g->d_den);
        hipFree((void *)g->d_den_idx);

        ckd_free((void *)g);
    }
#endif
}

gauden_dev_t *gauden_dev_copy(uint32 block_size, vector_t **feature, uint32 n_obs, model_inventory_t *inv, state_t *state_seq, uint32 n_state) {

    gauden_dev_t *g;
    uint32 *buf;
    uint32 s;
    
#ifndef GAUDEN_DEV
    return NULL;
#endif
    
    g = (gauden_dev_t *)ckd_calloc(1, sizeof(gauden_dev_t));
    
    g->n_feat = inv->gauden->n_feat;
    g->n_mgau = inv->gauden->n_mgau;
    g->n_density = inv->gauden->n_density;
    g->n_top = inv->gauden->n_top;
    g->n_cb_inverse = inv->n_cb_inverse;
    g->n_active_state = 0;  /* computed later */
    
    hipMalloc(&g->d_veclen, g->n_feat * sizeof(uint32));
    hipMalloc(&g->d_norm, g->n_mgau * g->n_feat * g->n_density * sizeof(float32));
    
    hipMalloc(&g->d_cb, n_state * sizeof(uint32));
    hipMalloc(&g->d_l_cb, n_state * sizeof(uint32));
    hipMalloc(&g->d_active_states, n_state * sizeof(uint32));
    
    g->d_mean_buflen = inv->gauden->mean[0][0][g->n_mgau * g->n_feat * g->n_density - 1] - inv->gauden->mean[0][0][0] + inv->gauden->veclen[g->n_feat - 1];
    hipMalloc(&g->d_mean_idx, g->n_mgau * g->n_feat * g->n_density * sizeof(float *));
    hipMalloc(&g->d_mean_buf, g->d_mean_buflen * sizeof(float));

    g->d_var_buflen = inv->gauden->var[0][0][g->n_mgau * g->n_feat * g->n_density - 1] - inv->gauden->var[0][0][0] + inv->gauden->veclen[g->n_feat - 1];
    hipMalloc(&g->d_var_idx, g->n_mgau * g->n_feat * g->n_density * sizeof(float *));
    hipMalloc(&g->d_var_buf, g->d_var_buflen * sizeof(float));

    CUDA_SAFE_CALL(hipMalloc(&g->d_den, (block_size + 1) * g->n_cb_inverse * g->n_feat * g->n_top * sizeof(float64)));
    CUDA_SAFE_CALL(hipMalloc(&g->d_den_idx, (block_size + 1) * g->n_cb_inverse * g->n_feat * g->n_top * sizeof(uint32)));

    g->d_feature_buflen = feature[0][n_obs * g->n_feat - 1] - feature[0][0] + inv->gauden->veclen[g->n_feat - 1];
    CUDA_SAFE_CALL(hipMalloc(&g->d_feature_idx, n_obs * g->n_feat * sizeof(float *)));
    CUDA_SAFE_CALL(hipMalloc(&g->d_feature_buf, g->d_feature_buflen * sizeof(float)));

    CUDA_SAFE_CALL(hipMemcpy(g->d_feature_idx, feature[0], n_obs * g->n_feat * sizeof(float *), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(g->d_feature_buf, feature[0][0], g->d_feature_buflen * sizeof(float), hipMemcpyHostToDevice));
    
    /* veclen, norm, den, den_idx */
    hipMemcpy(g->d_veclen, inv->gauden->veclen, g->n_feat * sizeof(uint32), hipMemcpyHostToDevice);
    hipMemcpy(g->d_norm, inv->gauden->norm[0][0], g->n_mgau * g->n_feat * g->n_density * sizeof(float32), hipMemcpyHostToDevice);
    
    /* state_seq -> d_cb, d_l_cb, d_mixw */
    buf = (uint32 *)ckd_calloc(n_state, sizeof(uint32));
    for (s = 0; s < n_state; s++) buf[s] = state_seq[s].cb;
    hipMemcpy(g->d_cb, buf, n_state * sizeof(uint32), hipMemcpyHostToDevice);
    for (s = 0; s < n_state; s++) buf[s] = state_seq[s].l_cb;
    hipMemcpy(g->d_l_cb, buf, n_state * sizeof(uint32), hipMemcpyHostToDevice);
/*    for (s = 0; s < g->n_state; s++) buf[s] = state_seq[s].mixw;*/
    for (s = 0; s < n_state; s++) {
        if (state_seq[s].mixw != TYING_NON_EMITTING) {
            buf[g->n_active_state++] = s;
        }
    }
    hipMemcpy(g->d_active_states, buf, g->n_active_state * sizeof(uint32), hipMemcpyHostToDevice);
    ckd_free((void *)buf);
    
    /* mean, var, feature */
    hipMemcpy(g->d_mean_idx, inv->gauden->mean[0][0], g->n_mgau * g->n_feat * g->n_density * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(g->d_mean_buf, inv->gauden->mean[0][0][0], g->d_mean_buflen * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g->d_var_idx, inv->gauden->var[0][0], g->n_mgau * g->n_feat * g->n_density * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(g->d_var_buf, inv->gauden->var[0][0][0], g->d_var_buflen * sizeof(float), hipMemcpyHostToDevice);

    return g;
}


/*
__global__ void device_init_3d_kernel(char *mem, char ***ref1, char **ref2, size_t elemsize, size_t d1, size_t d2, size_t d3);
__global__ void device_init_4d_kernel(char *mem, char ****ref1, char ***ref2, char **ref3, size_t elemsize, size_t d1, size_t d2, size_t d3, size_t d4);


void *
device_alloc_3d(size_t d1, size_t d2, size_t d3, size_t elemsize)
{
    dim3 bdim(16, 1, 1);
    dim3 gdim(ceil(d1 / (float)bdim.x), 1, 1);

    char ***ref1, **ref2, *mem;

    hipMalloc(&mem, d1 * d2 * d3 * elemsize);
    hipMalloc(&ref2, d1 * d2 * sizeof(void *));
    hipMalloc(&ref1, d1 * sizeof(void **));

    device_init_3d_kernel<<<gdim, bdim>>>(mem, ref1, ref2, elemsize, d1, d2, d3);

    return ref1;
}


void *
device_alloc_4d(size_t d1,
		  size_t d2,
		  size_t d3,
		  size_t d4,
		  size_t elemsize)
{
    dim3 bdim(16, 1, 1);
    dim3 gdim(ceil(d1 / (float)bdim.x), 1, 1);

    char *mem;
    char ****ref1;
    char ***ref2;
    char **ref3;

    hipMalloc(&mem, d1 * d2 * d3 * d4 * elemsize);
    hipMalloc(&ref3, d1 * d2 * d3 * sizeof(void *));
    hipMalloc(&ref2, d1 * d2 * sizeof(void **));
    hipMalloc(&ref1, d1 * sizeof(void ***));
    
    device_init_4d_kernel<<<gdim, bdim>>>(mem, ref1, ref2, ref3, elemsize, d1, d2, d3, d4);
    
    return ref1;
}


void
device_free_3d(void *inptr) {
    void ***ptr = (void ***)inptr;
    if (ptr == NULL)
        return;
    hipFree(ptr[0][0]);
    hipFree(ptr[0]);
    hipFree(ptr);
}


void
device_free_4d(void *inptr) {
    void ****ptr = (void ****)inptr;
    if (ptr == NULL)
        return;
    hipFree(ptr[0][0][0]);
    hipFree(ptr[0][0]);
    hipFree(ptr[0]);
    hipFree(ptr);
}


__global__ void device_init_3d_kernel(char *mem, char ***ref1, char **ref2, size_t elemsize, size_t d1, size_t d2, size_t d3) {
    int i, j;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d1) return;
    
    ref1[i] = ref2 + i * d2;
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        ref1[i][j] = mem + (i * d2 + j) * d3 * elemsize;
    }
}


__global__ void device_init_4d_kernel(char *mem, char ****ref1, char ***ref2, char **ref3, size_t elemsize, size_t d1, size_t d2, size_t d3, size_t d4) {
    int i, j, k;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d1) return;
    
    ref1[i] = ref2 + i * d2;
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        ref1[i][j] = ref3 + (i * d2 + j) * d3;
    }
    
    __syncthreads();
    
    for (j = 0; j < d2; j++) {
        for (k = 0; k < d3; k++) {
            ref1[i][j][k] = mem + ((i * d2 + j) * d3 + k) * d4 * elemsize;
        }
    }
}
*/

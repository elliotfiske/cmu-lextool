#include "hip/hip_runtime.h"
/* -*- c-basic-offset: 4 -*- */
/* ====================================================================
 * Copyright (c) 1995-2000 Carnegie Mellon University.  All rights 
 * reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer. 
 *
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in
 *    the documentation and/or other materials provided with the
 *    distribution.
 *
 * This work was supported in part by funding from the Defense Advanced 
 * Research Projects Agency and the National Science Foundation of the 
 * United States of America, and the CMU Sphinx Speech Consortium.
 *
 * THIS SOFTWARE IS PROVIDED BY CARNEGIE MELLON UNIVERSITY ``AS IS'' AND 
 * ANY EXPRESSED OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, 
 * THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL CARNEGIE MELLON UNIVERSITY
 * NOR ITS EMPLOYEES BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, 
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY 
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE 
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * ====================================================================
 *
 */
/*********************************************************************
 *
 * File: forward.c
 * 
 * Description: 
 * 
 * Author: 
 *         Eric H. Thayer (eht@cs.cmu.edu)
 *********************************************************************/

#include <s3/model_inventory.h>
#include <s3/s3phseg_io.h>
#include <s3/vector.h>
#include <sphinxbase/ckd_alloc.h>
#include <s3/gauden.h>
#include <s3/state.h>
#include <s3/s3.h>

#include <s3/profile.h>

#include <assert.h>
#include <math.h>
#include <string.h>

#include "forward.h"

#define FORWARD_DEBUG 0
#define INACTIVE        0xffff

void
forward_init_arrays(
        float64 ***active_alpha,
        uint32 ***active_astate,
        uint32 **n_active_astate,
        uint32 ***bp,
        float64 **scale,
        float64 ***dscale,
        uint32 n_obs)
{
    *active_alpha = (float64 **)ckd_calloc(n_obs, sizeof(float64 *));
    *active_astate = (uint32 **)ckd_calloc(n_obs, sizeof(uint32 *));
    *n_active_astate = (uint32 *)ckd_calloc(n_obs, sizeof(uint32));
    *scale = (float64 *)ckd_calloc(n_obs, sizeof(float64));
    *dscale = (float64 **)ckd_calloc(n_obs, sizeof(float64 *));
    if (bp) {
        *bp = (uint32 **)ckd_calloc(n_obs, sizeof(uint32 *));
    }
    memset(*active_alpha, 0, n_obs * sizeof(float64 *));
    memset(*active_astate, 0, n_obs * sizeof(uint32 *));
    memset(*n_active_astate, 0, n_obs * sizeof(uint32));
    memset(*scale, 0, n_obs * sizeof(float64));
    memset(*dscale, 0, n_obs * sizeof(float64 *));
    if (bp) {
        memset(*bp, 0, n_obs * sizeof(uint32 *));
    }
}

void forward_free_arrays(
        float64 ***active_alpha,
        uint32 ***active_astate,
        uint32 **n_active_astate,
        uint32 ***bp,
        float64 **scale,
        float64 ***dscale)
{
    ckd_free(*active_alpha);
    ckd_free(*active_astate);
    ckd_free(*n_active_astate);
    ckd_free(*scale);
    ckd_free(*dscale);
    if (bp) {
        ckd_free(*bp);
    }

    *active_alpha = NULL;
    *active_astate = NULL;
    *n_active_astate = NULL;
    *scale = NULL;
    *dscale = NULL;
    if (bp) {
        *bp = NULL;
    }
}

void
forward_clear_arrays(
        float64 **active_alpha,
        uint32 **active_astate,
        uint32 **bp,
        float64 **dscale,
        uint32 n_obs)
{
    uint32 t;
    
    for (t = 0; t < n_obs; t++) {
        ckd_free(active_alpha[t]);
        ckd_free(active_astate[t]);
        ckd_free(dscale[t]);
        if (bp) {
            ckd_free(bp[t]);
        }
    }
}



/*********************************************************************
 *
 * Function: 
 *         forward
 *
 * Description: 
 *         This function computes the scaled forward variable, alpha.
 *
 *        In order to conserve memory for long utterances, only the
 *        active (i.e. non-zero) alpha values are stored.
 *
 * Function Inputs: 
 *         float64 **active_alpha -
 *                On the successful return of this function,
 *                this array contains the scaled alpha variable for
 *                active states for all input observations.  For any
 *                input observation at time t (t >= 0  && t < n_obs), the number
 *                of active states can be found by accessing
 *                n_active_astate[t].  So for some time t, the active
 *                scaled alpha values are active_alpha[t][i] where
 *                i >= 0 and i < n_active_astate[t].  For some valid t and
 *                i, the sentence HMM state id can be found by accessing
 *                active_astate[t][i].
 *
 *         uint32 **active_astate -
 *                On the successful return of this function,
 *                this array contains the mapping of active state indexes
 *                into sentence HMM indexes (i.e. indexes into the state_seq[]
 *                array).  The active states of the sentence HMM may be
 *                enumerated by accessing active_astate[t][i] over all
 *                t (t >= 0 && t < n_obs) and i
 *                (i >= 0 && i < n_active_astate[t]).
 *
 *         uint32 *n_active_astate -
 *                On the successful return of this function,
 *                this array contains the number of active states for all
 *                t (t >= 0 && t < n_obs).
 *
 *        float64 *scale -
 *                On the successful return of this function,
 *                 this variable contains the scale factor applied to the
 *                alpha variable for all t (t >= 0 && t < n_obs).
 *
 *        vector_t **feature -
 *                This variable contains the input observation vectors.
 *                The value feature[t][f] must be valid for
 *                t >= 0 && t < n_obs and f >= 0 && f < n_feat) where
 *                n_feat is the # of assumed statistically independent
 *                feature streams to be modelled.
 *
 *         uint32 **bp -
 *                On the successful return of this function,
 *                this array contains backtrace pointers for active states
 *              for all input observations except the first timepoint.
 *              As in active_alpha[], the sentence HMM state id can be
 *              found by accessing active_astate[t][i].  NOTE!
 *              This is a "raw" backpointer array and as such, it contains
 *              pointers to non-emitting states.  These pointers refer to
 *              the current frame rather than the previous one.  Thus,
 *              the state id of the backpointer is either
 *              active_astate[t-1][bp[t][i]] (for normal states) or
 *              active_astate[t][bp[t][i]] (for non-emitting states).
 *
 *        uint32 n_obs -
 *                This variable contains the number of input observation
 *                vectors seen given the model.
 *
 *        state_t *state_seq -
 *                This is a list of model state structures which define
 *                the sentence HMM for this observation sequence.
 *
 *        uint32 n_state -
 *                The total # of states in the sentence HMM for this
 *                utterance.
 *
 *        model_inventory_t *inv -
 *                This structure contains the inventory of initial acoustic
 *                model parameters.
 *
 *        float64 beam -
 *                A pruning beam to apply to the evaluation of the alpha
 *                variable.
 *
 *      s3phseg_t *phseg -
 *              An optional phone segmentation to use to constrain the
 *              forward lattice.
 *
 * Global Inputs: 
 *         None
 *
 * Return Values: 
 *
 *         S3_SUCCESS
 *                The alpha variable was completed successfully for this
 *                observation sequence and model.
 *
 *        S3_ERROR
 *                Some error was detected that prevented the computation of the
 *                variable.
 *
 * Global Outputs: 
 *         None
 *
 * Errors: 
 *         - Initial alpha value < epsilon
 *        - If semi-continuous models, unable to normalize input frame
 *        - Output liklihood underflow
 *        - Alpha variable < epsilon for all active states
 *
 *********************************************************************/


int32
forward(float64 **active_alpha,
        uint32 **active_astate,
        uint32 *n_active_astate,
        uint32 **bp,
        float64 *scale,
        float64 **dscale,
        vector_t **feature,
        uint32 n_obs,
        state_t *state_seq,
        uint32 n_state,
        model_inventory_t *inv,
        float64 beam,
        s3phseg_t *phseg,
        uint32 mmi_train)
{
    int32 retval = S3_SUCCESS;
        
    float64 **red_active_alpha;
    uint32 **red_active_astate;
    uint32 *red_n_active_astate;
    uint32 **red_bp = NULL;
    float64 *red_scale;
    float64 **red_dscale;
    
    uint32 block_size = 11;
    uint32 n_red = ceil(n_obs / (float64)block_size);
    int32 t;    /* should be signed! iterating backwards to 0 and checking by >= 0 */

    /*
     * Allocate space for the initial state in the alpha
     * and active state arrays
     * Allocate the bestscore array for embedded Viterbi
     */
    
    forward_init_arrays(&red_active_alpha, &red_active_astate, &red_n_active_astate, &red_bp, &red_scale, &red_dscale, n_red);
    
    retval = forward_reduced(
            red_active_alpha, red_active_astate, red_n_active_astate, red_bp, red_scale, red_dscale,
            feature, block_size, n_obs, state_seq, n_state, inv, beam, phseg, mmi_train);
    if (retval != S3_SUCCESS) {
        goto cleanup;
    }
    
    for (t = n_red - 1; t >= 0; t--) {
        retval = forward_recompute(
            active_alpha + (t * block_size), active_astate + (t * block_size), n_active_astate + (t * block_size),
            bp + (t * block_size), scale + (t * block_size), dscale + (t * block_size),
            red_active_alpha, red_active_astate, red_n_active_astate, red_bp, red_scale, red_dscale,
            feature, t, block_size, n_obs, state_seq, n_state, inv, beam, phseg, mmi_train);
            
        if (retval != S3_SUCCESS) {
            goto cleanup;
        }
        
    }
    
cleanup:
    forward_clear_arrays(red_active_alpha, red_active_astate, red_bp, red_dscale, n_red);
    forward_free_arrays(&red_active_alpha, &red_active_astate, &red_n_active_astate, &red_bp, &red_scale, &red_dscale);

    return retval;
}


int32
forward_recompute(float64 **loc_active_alpha,
        uint32 **loc_active_astate,
        uint32 *loc_n_active_astate,
        uint32 **loc_bp,
        float64 *loc_scale,
        float64 **loc_dscale,
        float64 **red_active_alpha,
        uint32 **red_active_astate,
        uint32 *red_n_active_astate,
        uint32 **red_bp,
        float64 *red_scale,
        float64 **red_dscale,
        vector_t **feature,
        uint32 block_idx,
        uint32 block_size,
        uint32 n_obs,
        state_t *state_seq,
        uint32 n_state,
        model_inventory_t *inv,
        float64 beam,
        s3phseg_t *phseg,
        uint32 mmi_train)
{
    int32 retval = S3_SUCCESS;

    uint32 block_obs = block_size;
    
    if (block_idx * block_size + block_obs > n_obs) {
        block_obs = n_obs - block_idx * block_size;
    }
    
    loc_n_active_astate[0] = red_n_active_astate[block_idx];
    
    loc_active_alpha[0] = (float64 *)ckd_calloc(loc_n_active_astate[0], sizeof(float64));
    memcpy(loc_active_alpha[0], red_active_alpha[block_idx], loc_n_active_astate[0] * sizeof(float64));
    
    loc_active_astate[0] = (uint32 *)ckd_calloc(loc_n_active_astate[0], sizeof(uint32));
    memcpy(loc_active_astate[0], red_active_astate[block_idx], loc_n_active_astate[0] * sizeof(uint32));
    
    loc_scale[0] = red_scale[block_idx];
    
    loc_dscale[0] = (float64 *)ckd_calloc(inv->gauden->n_feat, sizeof(float64));
    memcpy(loc_dscale[0], red_dscale[block_idx], inv->gauden->n_feat * sizeof(float64));
    
    if (loc_bp) {
        loc_bp[0] = (uint32 *)ckd_calloc(loc_n_active_astate[0], sizeof(uint32));
        memcpy(loc_bp[0], red_bp[block_idx], loc_n_active_astate[0] * sizeof(uint32));
    }
    
    retval = forward_local(
        loc_active_alpha, loc_active_astate, loc_n_active_astate, loc_bp, loc_scale, loc_dscale,
        feature + (block_idx * block_size), block_obs, state_seq, n_state, inv, beam, phseg, mmi_train, (block_idx * block_size));
    
    return retval;
}

int32
forward_reduced(float64 **active_alpha,
        uint32 **active_astate,
        uint32 *n_active_astate,
        uint32 **bp,
        float64 *scale,
        float64 **dscale,
        vector_t **feature,
        uint32 block_size,
        uint32 n_obs,
        state_t *state_seq,
        uint32 n_state,
        model_inventory_t *inv,
        float64 beam,
        s3phseg_t *phseg,
        uint32 mmi_train)
{
    int32 retval = S3_SUCCESS;
        
    float64 **loc_active_alpha;
    uint32 **loc_active_astate;
    uint32 *loc_n_active_astate;
    uint32 **loc_bp = NULL;
    float64 *loc_scale;
    float64 **loc_dscale;
    
    uint32 n_red = ceil(n_obs / (float64)block_size);
    int t;

    /*
     * Allocate space for the initial state in the alpha
     * and active state arrays
     * Allocate the bestscore array for embedded Viterbi
     */
    
    forward_init_arrays(&loc_active_alpha, &loc_active_astate, &loc_n_active_astate, &loc_bp, &loc_scale, &loc_dscale, block_size + 1);
    
    loc_active_alpha[0] = (float64 *)ckd_calloc(1, sizeof(float64));
    loc_active_astate[0] = (uint32 *)ckd_calloc(1, sizeof(uint32));
    loc_n_active_astate[0] = 1;

    loc_active_alpha[0][0] = 1.0;
    loc_active_astate[0][0] = 0;
    
    if (bp) {
        loc_bp[0] = (uint32 *)ckd_calloc(1, sizeof(uint32));
    }
    
    for (t = 0; t < n_red; t++) {
        uint32 block_obs = block_size + 1;
        
        if (t * block_size + block_obs > n_obs) {
            block_obs = n_obs - t * block_size;
        }

        retval = forward_local(
            loc_active_alpha, loc_active_astate, loc_n_active_astate, loc_bp, loc_scale, loc_dscale,
            feature + (t * block_size), block_obs, state_seq, n_state, inv, beam, phseg, mmi_train, (t * block_size));
        if (retval != S3_SUCCESS) {
            goto cleanup;
        }
        
        n_active_astate[t] = loc_n_active_astate[0];

        active_alpha[t] = (float64 *)ckd_calloc(n_active_astate[t], sizeof(float64));
        memcpy(active_alpha[t], loc_active_alpha[0], n_active_astate[t] * sizeof(float64));

        active_astate[t] = (uint32 *)ckd_calloc(n_active_astate[t], sizeof(uint32));
        memcpy(active_astate[t], loc_active_astate[0], n_active_astate[t] * sizeof(uint32));

        scale[t] = loc_scale[0];

        dscale[t] = (float64 *)ckd_calloc(inv->gauden->n_feat, sizeof(float64));
        memcpy(dscale[t], loc_dscale[0], inv->gauden->n_feat * sizeof(float64));
        if (bp) {
            bp[t] = (uint32 *)ckd_calloc(n_active_astate[t], sizeof(uint32));
            memcpy(bp[t], loc_bp[0], n_active_astate[t] * sizeof(uint32));
        }
        
        if (t < n_red - 1) {
            loc_n_active_astate[0] = loc_n_active_astate[block_size];
            
            loc_active_alpha[0] = (float64 *)ckd_realloc(loc_active_alpha[0], loc_n_active_astate[0] * sizeof(float64));
            memcpy(loc_active_alpha[0], loc_active_alpha[block_size], loc_n_active_astate[0] * sizeof(float64));
            
            loc_active_astate[0] = (uint32 *)ckd_realloc(loc_active_astate[0], loc_n_active_astate[0] * sizeof(uint32));
            memcpy(loc_active_astate[0], loc_active_astate[block_size], loc_n_active_astate[0] * sizeof(uint32));
            
            loc_scale[0] = loc_scale[block_size];
            
            memcpy(loc_dscale[0], loc_dscale[block_size], inv->gauden->n_feat * sizeof(float64));
            if (bp) {
                loc_bp[0] = (uint32 *)ckd_realloc(loc_bp[0], loc_n_active_astate[0] * sizeof(uint32));
                memcpy(loc_bp[0], loc_bp[block_size], loc_n_active_astate[0] * sizeof(uint32));
            }
        }
        forward_clear_arrays(loc_active_alpha + 1, loc_active_astate + 1, (bp ? (loc_bp + 1) : NULL), loc_dscale + 1, block_obs - 1);
    }
    forward_clear_arrays(loc_active_alpha, loc_active_astate, (bp ? loc_bp : NULL), loc_dscale, 1);
    
cleanup:
    forward_free_arrays(&loc_active_alpha, &loc_active_astate, &loc_n_active_astate, &loc_bp, &loc_scale, &loc_dscale);

    return retval;
}

/*#include <sys/time.h>

void startTimer(struct timeval *timer){
        gettimeofday(timer, NULL);
}

int stopTimer(struct timeval *timer){
        struct timeval tmp;
        gettimeofday(&tmp, NULL);
        tmp.tv_sec -= timer->tv_sec;
        tmp.tv_usec -= timer->tv_usec;
        if (tmp.tv_usec < 0){
                tmp.tv_usec+=1000000;
                tmp.tv_sec--;
        }
        return (int)(tmp.tv_usec + tmp.tv_sec*1000000);
}*/


int32
forward_local(float64 **active_alpha,
        uint32 **active_astate,
        uint32 *n_active_astate,
        uint32 **bp,
        float64 *scale,
        float64 **dscale,
        vector_t **feature,
        uint32 n_obs,
        state_t *state_seq,
        uint32 n_state,
        model_inventory_t *inv,
        float64 beam,
        s3phseg_t *phseg,
        uint32 mmi_train,
        uint32 t_offset)
{
    int32 retval = S3_SUCCESS;
    
    uint32 *next_active = (uint32 *)ckd_calloc(n_state, sizeof(uint32));
    uint32 *active_l_cb = (uint32 *)ckd_calloc(n_state, sizeof(uint32));
    uint16 *amap = (uint16 *)ckd_calloc(n_state, sizeof(uint16));
    uint32 *acbframe = (uint32 *)ckd_calloc(inv->n_cb_inverse, sizeof(uint32));

    float64 ****now_den = (float64 ****)ckd_calloc_4d(n_obs, inv->n_cb_inverse, gauden_n_feat(inv->gauden), gauden_n_top(inv->gauden),
                                         sizeof(float64));
    uint32 ****now_den_idx = (uint32 ****)ckd_calloc_4d(n_obs, inv->n_cb_inverse, gauden_n_feat(inv->gauden), gauden_n_top(inv->gauden),
                                            sizeof(uint32));

    float64 *best_pred = (float64 *)ckd_calloc(1, sizeof(float64));
    uint32 aalpha_alloc = n_active_astate[0];
    
    float64 outprob_0;
    uint32 t, i;

    if (t_offset == 0) {
        /* Compute the component Gaussians for state 0 mixture density */
        gauden_compute_log(now_den[0][state_seq[0].l_cb],
                       now_den_idx[0][state_seq[0].l_cb],
                       feature[0],
                       inv->gauden,
                       state_seq[0].cb, NULL);

        active_l_cb[0] = state_seq[0].l_cb;
        acbframe[state_seq[0].l_cb] = 0;
                       
        dscale[0] = gauden_scale_densities_fwd(now_den[0], now_den_idx[0],
                                           active_l_cb, 1, inv->gauden);

        /* Compute the mixture density value for state 0 time 0 */
        outprob_0 = gauden_mixture(now_den[0][state_seq[0].l_cb],
                                now_den_idx[0][state_seq[0].l_cb],
                                inv->mixw[state_seq[0].mixw],
                                inv->gauden);
        if (outprob_0 <= MIN_IEEE_NORM_POS_FLOAT32) {
            E_ERROR("Small output prob (== %.2e) seen at frame 0 state 0\n", outprob_0);
            retval = S3_ERROR;
            goto cleanup;
        }
        /* Compute scale for t == 0 */
        scale[0] = 1.0 / outprob_0;
    }
    
/*    struct timeval timer;
    startTimer(&timer);*/
    
    for (t = 1; t < n_obs; t++) {
        for (i = 0; i < n_state; i++) {
            if (state_seq[i].mixw != TYING_NON_EMITTING) {
                uint32 l_cb = state_seq[i].l_cb;

                gauden_compute_log(now_den[t][l_cb], now_den_idx[t][l_cb],
                   feature[t], inv->gauden, state_seq[i].cb, ((inv->n_cb_inverse == 1) ? now_den_idx[t-1][l_cb] : NULL));
                           /* Preinitializing topn only really makes a difference 
                              for semi-continuous (inv->n_cb_inverse == 1) models. */
            }
        }
    }
    
/*    int tm = stopTimer(&timer);
    E_INFO("MICHAL: n_obs=%u n_state=%u time=%u\n", n_obs, n_state, tm);*/

    /* Initialize the active state map such that all states are inactive */
    for (i = 0; i < n_state; i++) {
        amap[i] = INACTIVE;
    }
    
    if (bp) {
        best_pred = (float64 *)ckd_calloc(aalpha_alloc, sizeof(float64));
    }
    
    /* Compute scaled alpha over all remaining time in the utterance */
    for (t = 1; t < n_obs; t++) {
    
        uint32 n_next_active = 0;
        uint32 n_active_l_cb = 0;
        int can_prune_phseg = 0;
        float64 balpha = 0.0;
        uint32 i, j, s, u;

        /* assume next active state set about the same size as current;
           adjust to actual size as necessary later */
        active_alpha[t] = (float64 *)ckd_calloc(aalpha_alloc, sizeof(float64));
        if (bp) {
            bp[t] = (uint32 *)ckd_calloc(aalpha_alloc, sizeof(uint32));
            /* reallocate the best score array and zero it out */
            best_pred = (float64 *)ckd_realloc(best_pred, aalpha_alloc * sizeof(float64));
            memset(best_pred, 0, aalpha_alloc * sizeof(float64));
        }

        /* For all active states at the previous frame, activate their
           successors in this frame and compute codebooks. */
        /* (these are pre-computed so they can be scaled to avoid underflows) */
        for (s = 0; s < n_active_astate[t-1]; s++) {
            i = active_astate[t-1][s];
            /* get list of states adjacent to active state i */
            /* activate them all, computing their codebook densities if necessary */
            for (u = 0; u < state_seq[i].n_next; u++) {
            
                j = state_seq[i].next_state[u];

                if (state_seq[j].mixw != TYING_NON_EMITTING) {
                    if (amap[j] == INACTIVE) {
                        uint32 l_cb = state_seq[j].l_cb;
                        
                        if (acbframe[l_cb] != t) {
                            /* Component density values not yet computed */
                            active_l_cb[n_active_l_cb++] = l_cb;
                            acbframe[l_cb] = t;
                        }

                        /* Initialize the alpha variable to zero */
                        active_alpha[t][n_next_active] = 0;

                        /* Put next state j into the active list */
                        /* Map active state list index to sentence HMM index */
                        amap[j] = n_next_active;
                        next_active[n_next_active] = j;

                        ++n_next_active;

                        if (n_next_active == aalpha_alloc) {
                            /* Need to reallocate the active_alpha array */
                            aalpha_alloc *= 2;
                            active_alpha[t] = (float64 *)ckd_realloc(active_alpha[t],
                                                          sizeof(float64) * aalpha_alloc);
                            
                            if (bp) {
                                bp[t] = (uint32 *)ckd_realloc(bp[t],
                                                    sizeof(uint32) * aalpha_alloc);
                                best_pred = (float64 *)ckd_realloc(best_pred,
                                                                   sizeof(float64) * aalpha_alloc);
                                memset(bp[t] + aalpha_alloc / 2,
                                       0, sizeof(uint32) * (aalpha_alloc / 2));
                                memset(best_pred + aalpha_alloc / 2,
                                       0, sizeof(float64) * (aalpha_alloc / 2));
                            }
                        }
                    }
                }
            }
        }

        /* Cope w/ numerical issues by dividing densities by max density */
        dscale[t] = gauden_scale_densities_fwd(now_den[t], now_den_idx[t],
                                               active_l_cb, n_active_l_cb, inv->gauden);
        
        /* Now, for all active states in the previous frame, compute
           alpha for all successors in this frame. */
        for (s = 0; s < n_active_astate[t-1]; s++) {
            i = active_astate[t-1][s];

            /* For all emitting states j adjacent to i, update their
             * alpha values.  */
            for (u = 0; u < state_seq[i].n_next; u++) {
                j = state_seq[i].next_state[u];
                uint32 l_cb = state_seq[j].l_cb;

                if (state_seq[j].mixw != TYING_NON_EMITTING) {
                    /* Next state j is an emitting state */
                    float64 outprob_j = gauden_mixture(now_den[t][l_cb],
                                        now_den_idx[t][l_cb],
                                        inv->mixw[state_seq[j].mixw],
                                        inv->gauden);
                    /* update backpointers bp[t][j] */
                    float64 x = active_alpha[t-1][s] * state_seq[i].next_tprob[u];
                    if (bp) {
                        if (x > best_pred[amap[j]]) {
                            best_pred[amap[j]] = x;
                            bp[t][amap[j]] = s;
                        }
                    }
                    
                    /* update the unscaled alpha[t][j] */
                    active_alpha[t][amap[j]] += x * outprob_j;
                }
                else {
                    /* already done below in the prior time frame */
                }
            }
        }

        /* Now, for all active states in this frame, consume any
           following non-emitting states (multiplying in their
           transition probabilities)  */
        for (s = 0; s < n_next_active; s++) {
            i = next_active[s];

            for (u = 0; u < state_seq[i].n_next; u++) {
                j = state_seq[i].next_state[u];
                /* for any non-emitting ones */
                if (state_seq[j].mixw == TYING_NON_EMITTING) {
                    float64 x = active_alpha[t][s] * state_seq[i].next_tprob[u];

                    /* activate this state if necessary */
                    if (amap[j] == INACTIVE) {
                        active_alpha[t][n_next_active] = 0;
                        amap[j] = n_next_active;
                        next_active[n_next_active] = j;
                        ++n_next_active;

                        if (n_next_active == aalpha_alloc) {
                            aalpha_alloc *= 2;
                            active_alpha[t] = (float64 *)ckd_realloc(active_alpha[t],
                                                          sizeof(float64) * aalpha_alloc);
                            
                            if (bp) {
                                bp[t] = (uint32 *)ckd_realloc(bp[t],
                                                    sizeof(uint32) * aalpha_alloc);
                                best_pred = (float64 *)ckd_realloc(best_pred,
                                                                   sizeof(float64) * aalpha_alloc);
                                memset(bp[t] + aalpha_alloc / 2,
                                       0, sizeof(uint32) * (aalpha_alloc / 2));
                                memset(best_pred + aalpha_alloc / 2,
                                       0, sizeof(float64) * (aalpha_alloc / 2));
                            }
                        }
                        if (bp) {
                            /* Give its backpointer a default value */
                            bp[t][amap[j]] = s;
                            best_pred[amap[j]] = x;
                        }
                    }

                    /* update backpointers bp[t][j] */
                    if (bp && x > best_pred[amap[j]]) {
                        bp[t][amap[j]] = s;
                        best_pred[amap[j]] = x;
                    }
                    /* update its alpha value */
                    active_alpha[t][amap[j]] += x;
                }
            }
        }

        /* find best alpha value in current frame for pruning and scaling purposes */
        /* also take the argmax to find the best backtrace */
        for (s = 0; s < n_next_active; s++) {
            if (balpha < active_alpha[t][s]) {
                balpha = active_alpha[t][s];
            }
        }

        /* cope with some pathological case */
        if (balpha == 0.0 && n_next_active > 0) {
            E_ERROR("All %u active states,", n_next_active);
            for (s = 0; s < n_next_active; s++) {
                if (state_seq[next_active[s]].mixw != TYING_NON_EMITTING)
                    fprintf(stderr, " %u", state_seq[next_active[s]].mixw);
                else
                    fprintf(stderr, " N(%u,%u)",
                            state_seq[next_active[s]].tmat, state_seq[next_active[s]].m_state);

            }
            fprintf(stderr, ", zero at time %u\n", t);
            fflush(stderr);
            retval = S3_ERROR;
            goto cleanup;
        }

        /* and some related pathological cases */
        if (balpha < 1e-300) {
            E_ERROR("Best alpha < 1e-300\n");
            retval = S3_ERROR;
            goto cleanup;
        }
        if (n_next_active == 0) {
            E_ERROR("No active states at time %u\n", t);
            retval = S3_ERROR;
            goto cleanup;
        }

        /* compute the scale factor */
        scale[t] = 1.0 / balpha;
    
        /* Determine if phone segmentation-based pruning would leave
         * us with an empty active list (that would be bad!) */
        if (phseg) {
            /* Find active phone for this timepoint. */
            /* Move the pointer forward if necessary. */
            while ((t + t_offset) > phseg->ef)
                phseg = phseg->next;
                
            for (s = 0; s < n_next_active; ++s) 
                if (acmod_set_base_phone(inv->mdef->acmod_set, state_seq[next_active[s]].phn)
                    == acmod_set_base_phone(inv->mdef->acmod_set, phseg->phone))
                    break;
            can_prune_phseg = (s != n_next_active);
        }
        
        /* Prune active states for the next frame and rescale their alphas. */
        n_active_astate[t] = 0;
        active_astate[t] = (uint32 *)ckd_calloc(n_next_active, sizeof(uint32));
        for (s = 0; s < n_next_active; s++) {
            /* "Snap" the backpointers for non-emitting states, so
               that they don't point to bogus indices (we will use
               amap to recover them). */
            if (bp && state_seq[next_active[s]].mixw == TYING_NON_EMITTING) {
                bp[t][s] = next_active[bp[t][s]];
            }
            /* If we have a phone segmentation, use it instead of the beam. */
            if (phseg && can_prune_phseg) {
                if (acmod_set_base_phone(inv->mdef->acmod_set, state_seq[next_active[s]].phn)
                    == acmod_set_base_phone(inv->mdef->acmod_set, phseg->phone)) {
                    active_alpha[t][n_active_astate[t]] = active_alpha[t][s] * scale[t];
                    active_astate[t][n_active_astate[t]] = next_active[s];
                    if (bp)
                        bp[t][n_active_astate[t]] = bp[t][s];
                    amap[next_active[s]] = n_active_astate[t];
                    n_active_astate[t]++;
                }
                else {
                    amap[next_active[s]] = INACTIVE;
                }
            }
            else {
                /* compute the pruning threshold based on the beam */
                float64 pthresh = ((log10(balpha) + log10(beam) > -300) ? (balpha * beam) : 1e-300);
        
                if (active_alpha[t][s] > pthresh) {
                    active_alpha[t][n_active_astate[t]] = active_alpha[t][s] * scale[t];
                    active_astate[t][n_active_astate[t]] = next_active[s];
                    if (bp)
                        bp[t][n_active_astate[t]] = bp[t][s];
                    amap[next_active[s]] = n_active_astate[t];
                    n_active_astate[t]++;
                }
                else {
                    amap[next_active[s]] = INACTIVE;
                }
            }
        }
        
        /* Now recover the backpointers for non-emitting states. */
        if (bp) {
                for (s = 0; s < n_active_astate[t]; ++s) {
                    if (state_seq[active_astate[t][s]].mixw == TYING_NON_EMITTING) {
                        bp[t][s] = amap[bp[t][s]];
                    }
                }
        }
        
        /* And finally deactive all states. */
        for (s = 0; s < n_active_astate[t]; ++s) {
            amap[active_astate[t][s]] = INACTIVE;
        }
    }
    
cleanup:
    ckd_free(next_active);
    ckd_free(amap);
    ckd_free(active_l_cb);

    ckd_free(best_pred);

    ckd_free(acbframe);

    ckd_free_4d((void ****)now_den);
    ckd_free_4d((void ****)now_den_idx);
    
    return retval;
}

